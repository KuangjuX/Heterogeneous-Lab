#include <stdio.h>
#include <hip/hip_runtime.h>

#include <time.h>
#include <sys/time.h>

const long double TimeConvert = 1e6;
#define BLOCK_NUM 32   //块数量
#define THREAD_NUM 256 // 每个块中的线程数
#define LOOP_N BLOCK_NUM * THREAD_NUM * 1000000

__global__ void leib_pi(double* g_sum) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    double tmp = 0;
    int flag = -1;
    int idx = bid * THREAD_NUM + tid;
    int start = idx * 100000 + 1;
    int end = start + 100000;
    for (int i = start; i < end; i++) {
        tmp += flag * (1./(2 * i + 1));
        flag = -flag;
    }
    g_sum[bid*THREAD_NUM+tid] = tmp;
}

int main(){
    double *h_sum, *g_sum;
    double pi_v = 1;

    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);
    // allocate host memory
    h_sum = (double*) malloc(sizeof(double) * BLOCK_NUM * THREAD_NUM);

    // Allocate device memory
    hipMalloc((void **)&g_sum, sizeof(double) * BLOCK_NUM * THREAD_NUM);

    // Execute kernels
    leib_pi<<<BLOCK_NUM,THREAD_NUM>>>(g_sum);

    // Transfer output from device memory to host
    hipMemcpy(h_sum, g_sum, sizeof(double)*BLOCK_NUM*THREAD_NUM, hipMemcpyDeviceToHost);

    for (int i = 0; i < BLOCK_NUM * THREAD_NUM; i++) {
        pi_v += h_sum[i];
    }

    gettimeofday(&end_time, NULL);
    long long int start, end;
    start = start_time.tv_sec * TimeConvert + start_time.tv_usec;
    end = end_time.tv_sec * TimeConvert + end_time.tv_usec;
    long double span_time;
    span_time = (end - start) / TimeConvert;
    printf("花费时间: %.6LFs.\n", span_time);

    printf("计算的 pi 值为： %.10f\n", pi_v*4);

    hipFree(g_sum);
    free(h_sum);
}